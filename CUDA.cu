
#include <hip/hip_runtime.h>
#include <stdio.h>

 
__global__ 
void update(float *ad, float *bd, int ny,int nx)
{
   int x, y;

   x = blockIdx.x;
   y = threadIdx.x;
   if(x > 0 && y > 0 && x < nx-1 && y < ny-1) 
   	bd[x*ny+y] = ad[x*ny+y] + (ad[(x+1)*ny+y] + ad[(x-1)*ny+y] - 2 * ad[x*ny+y])/10 + (ad[x*ny+(y+1)] + ad[x*ny+(y-1)] -  2 * ad[x*ny+y])/10;
}

extern "C" float updateGPU(float **arr1, float **arr2, int nx, int ny, int steps)
{ 
	float *ad,*bd,s[nx*ny], milli = 0.0;
	int i, j;
	size_t size = nx*ny*sizeof(float);
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	printf("= mpi_heat2D - CUDA Version =\nGrid size: X = %d, Y = %d, Time steps = %d\n",nx,ny,steps);

    for (i = 0;i<nx;i++)
    	for (j = 0;j<ny;j++){
    		s[i*ny+j] = arr1[i][j];
    	}

	hipMalloc( (void**)&ad, size); 
	hipMemcpy( ad, s, size, hipMemcpyHostToDevice );

    hipMalloc( (void**)&bd, size);

	dim3 threads_per_block(ny);
	dim3 num_blocks(nx,1);
	
	hipEventRecord(start);
	for(i = 0; i < steps; i++){
		if(i%2 == 0)
			update<<<num_blocks, threads_per_block>>>(ad, bd, ny,nx);
		else
			update<<<num_blocks, threads_per_block>>>(bd, ad, ny,nx);
	}
	hipEventRecord(stop);
	if (i%2 == 0)
		hipMemcpy( s, ad, size, hipMemcpyDeviceToHost );
	else
		hipMemcpy( s, bd, size, hipMemcpyDeviceToHost );

	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli, start, stop);

	printf("Time Elapsed is %2.6f seconds\n",milli/1000);	
 
	hipFree( ad );
	hipFree( bd );

	hipEventDestroy(start);
	hipEventDestroy(stop);

	for (i = 0;i<nx;i++)
    	for (j = 0;j<ny;j++)
    		 arr1[i][j] = s[i*ny+j];

	return 1;
}